#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * ---------------------------------------------------------------------------*
 * @brief KTruss implementation
 *
 * @file ktruss.cu
 * --------------------------------------------------------------------------*/

#include <cugraph/utilities/error.hpp>

#include "Static/KTruss/KTruss.cuh"
#include <Hornet.hpp>
#include <StandardAPI.hpp>
#include <cugraph/algorithms.hpp>

using namespace hornets_nest;

namespace cugraph {

namespace detail {

template <typename VT, typename ET, typename WT>
std::unique_ptr<legacy::GraphCOO<VT, ET, WT>> ktruss_subgraph_impl(
  legacy::GraphCOOView<VT, ET, WT> const& graph, int k, rmm::mr::device_memory_resource* mr)
{
  using HornetGraph = hornet::gpu::Hornet<VT>;
  using UpdatePtr   = hornet::BatchUpdatePtr<VT, hornet::EMPTY, hornet::DeviceType::DEVICE>;
  using Update      = hornet::gpu::BatchUpdate<VT>;
  hipStream_t stream{nullptr};
  UpdatePtr ptr(graph.number_of_edges, graph.src_indices, graph.dst_indices);
  Update batch(ptr);

  HornetGraph hnt(graph.number_of_vertices + 1);
  hnt.insert(batch);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to initialize graph");

  KTruss kt(hnt);

  kt.init();
  kt.reset();
  kt.createOffSetArray();
  // NOTE : These parameters will become obsolete once we move to the updated
  // algorithm (https://ieeexplore.ieee.org/document/8547581)
  kt.setInitParameters(4,      // Number of threads per block per list intersection
                       8,      // Number of intersections per block
                       2,      // log2(Number of threads)
                       64000,  // Total number of blocks launched
                       32);    // Thread block dimension
  kt.reset();
  kt.sortHornet();

  kt.runForK(k);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to run");

  auto out_graph = std::make_unique<legacy::GraphCOO<VT, ET, WT>>(
    graph.number_of_vertices, kt.getGraphEdgeCount(), graph.has_data(), stream, mr);

  kt.copyGraph(out_graph->src_indices(), out_graph->dst_indices());

  kt.release();
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to release");

  return out_graph;
}
template <typename VT, typename ET, typename WT>
std::unique_ptr<legacy::GraphCOO<VT, ET, WT>> weighted_ktruss_subgraph_impl(
  legacy::GraphCOOView<VT, ET, WT> const& graph, int k, rmm::mr::device_memory_resource* mr)
{
  using HornetGraph = hornet::gpu::Hornet<VT, hornet::EMPTY, hornet::TypeList<WT>>;
  using UpdatePtr   = hornet::BatchUpdatePtr<VT, hornet::TypeList<WT>, hornet::DeviceType::DEVICE>;
  using Update      = hornet::gpu::BatchUpdate<VT, hornet::TypeList<WT>>;
  hipStream_t stream{nullptr};
  UpdatePtr ptr(graph.number_of_edges, graph.src_indices, graph.dst_indices, graph.edge_data);
  Update batch(ptr);

  HornetGraph hnt(graph.number_of_vertices + 1);
  hnt.insert(batch);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to initialize graph");

  KTrussWeighted<WT> kt(hnt);

  kt.init();
  kt.reset();
  kt.createOffSetArray();
  // NOTE : These parameters will become obsolete once we move to the updated
  // algorithm (https://ieeexplore.ieee.org/document/8547581)
  kt.setInitParameters(4,      // Number of threads per block per list intersection
                       8,      // Number of intersections per block
                       2,      // log2(Number of threads)
                       64000,  // Total number of blocks launched
                       32);    // Thread block dimension
  kt.reset();
  kt.sortHornet();

  kt.runForK(k);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to run");

  auto out_graph = std::make_unique<legacy::GraphCOO<VT, ET, WT>>(
    graph.number_of_vertices, kt.getGraphEdgeCount(), graph.has_data(), stream, mr);

  kt.copyGraph(out_graph->src_indices(), out_graph->dst_indices(), out_graph->edge_data());

  kt.release();
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to release");

  return out_graph;
}


template <typename vertex_t>
std::tuple<rmm::device_uvector<vertex_t>,
           rmm::device_uvector<vertex_t>>
ktruss_subgraph_impl_(raft::handle_t const& handle,
                     raft::device_span<vertex_t> src,
                     raft::device_span<vertex_t> dst,
                     size_t number_of_vertices,
                     int k)
{

  using HornetGraph = hornet::gpu::Hornet<vertex_t>;
  using UpdatePtr   = hornet::BatchUpdatePtr<vertex_t, hornet::EMPTY, hornet::DeviceType::DEVICE>;
  using Update      = hornet::gpu::BatchUpdate<vertex_t>;

  HornetGraph hnt(number_of_vertices + 1);

  // NOTE: Should a constant pointer be passed for @src and @dst
  UpdatePtr ptr(static_cast<int>(src.size()), src.data(), dst.data());
  Update batch(ptr);

  hnt.insert(batch);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to initialize graph");

  KTruss kt(hnt);

  kt.init();
  kt.reset();
  kt.createOffSetArray();
  // NOTE : These parameters will become obsolete once we move to the updated
  // algorithm (https://ieeexplore.ieee.org/document/8547581)
  kt.setInitParameters(4,      // Number of threads per block per list intersection
                       8,      // Number of intersections per block
                       2,      // log2(Number of threads)
                       64000,  // Total number of blocks launched
                       32);    // Thread block dimension
  kt.reset();
  kt.sortHornet();

  kt.runForK(k);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to run");

  rmm::device_uvector<vertex_t> result_src(kt.getGraphEdgeCount(), handle.get_stream());
  rmm::device_uvector<vertex_t> result_dst(kt.getGraphEdgeCount(), handle.get_stream());
  //std::optional<rmm::device_uvector<weight_t>> result_wgt{std::nullopt};

  kt.copyGraph(result_src.data(), result_dst.data());
  

  kt.release();
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to release");

  return std::make_tuple(std::move(result_src),
                         std::move(result_dst));

}

template <typename vertex_t, typename weight_t>
std::tuple<rmm::device_uvector<vertex_t>,
           rmm::device_uvector<vertex_t>,
           std::optional<rmm::device_uvector<weight_t>>>
weighted_ktruss_subgraph_impl_(raft::handle_t const& handle,
                     raft::device_span<vertex_t> src,
                     raft::device_span<vertex_t> dst,
                     std::optional<raft::device_span<weight_t>> wgt,
                     size_t number_of_vertices,
                     int k)
{
  using HornetGraph = hornet::gpu::Hornet<vertex_t, hornet::EMPTY, hornet::TypeList<weight_t>>;
  using UpdatePtr   = hornet::BatchUpdatePtr<vertex_t, hornet::TypeList<weight_t>, hornet::DeviceType::DEVICE>;
  using Update      = hornet::gpu::BatchUpdate<vertex_t, hornet::TypeList<weight_t>>;

  HornetGraph hnt(number_of_vertices + 1);

  UpdatePtr ptr(static_cast<int>(src.size()), src.data(), dst.data(), wgt->data());
  Update batch(ptr);

  hnt.insert(batch);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to initialize graph");

  KTrussWeighted<weight_t> kt(hnt);

  kt.init();
  kt.reset();
  kt.createOffSetArray();
  // NOTE : These parameters will become obsolete once we move to the updated
  // algorithm (https://ieeexplore.ieee.org/document/8547581)
  kt.setInitParameters(4,      // Number of threads per block per list intersection
                       8,      // Number of intersections per block
                       2,      // log2(Number of threads)
                       64000,  // Total number of blocks launched
                       32);    // Thread block dimension
  kt.reset();
  kt.sortHornet();

  kt.runForK(k);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to run");

  rmm::device_uvector<vertex_t> result_src(kt.getGraphEdgeCount(), handle.get_stream());
  rmm::device_uvector<vertex_t> result_dst(kt.getGraphEdgeCount(), handle.get_stream());
  std::optional<rmm::device_uvector<weight_t>> result_wgt{std::nullopt};

  result_wgt = rmm::device_uvector<weight_t>(kt.getGraphEdgeCount(), handle.get_stream());
  kt.copyGraph(result_src.data(), result_dst.data(), result_wgt->data());

  kt.release();
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to release");

  return std::make_tuple(std::move(result_src),
                         std::move(result_dst),
                         std::move(result_wgt));
}

}  // namespace detail

template <typename VT, typename ET, typename WT>
std::unique_ptr<legacy::GraphCOO<VT, ET, WT>> k_truss_subgraph(
  legacy::GraphCOOView<VT, ET, WT> const& graph, int k, rmm::mr::device_memory_resource* mr)
{
  CUGRAPH_EXPECTS(graph.src_indices != nullptr, "Graph source indices cannot be a nullptr");
  CUGRAPH_EXPECTS(graph.dst_indices != nullptr, "Graph destination indices cannot be a nullptr");

  if (graph.edge_data == nullptr) {
    return detail::ktruss_subgraph_impl(graph, k, mr);
  } else {
    return detail::weighted_ktruss_subgraph_impl(graph, k, mr);
  }
}

template std::unique_ptr<legacy::GraphCOO<int32_t, int32_t, float>>
k_truss_subgraph<int, int, float>(legacy::GraphCOOView<int, int, float> const&,
                                  int,
                                  rmm::mr::device_memory_resource*);

template std::unique_ptr<legacy::GraphCOO<int32_t, int32_t, double>>
k_truss_subgraph<int, int, double>(legacy::GraphCOOView<int, int, double> const&,
                                   int,
                                   rmm::mr::device_memory_resource*);



template <typename vertex_t, typename weight_t>
std::tuple<rmm::device_uvector<vertex_t>,
           rmm::device_uvector<vertex_t>,
           std::optional<rmm::device_uvector<weight_t>>>
k_truss_subgraph(raft::handle_t const& handle,
                 raft::device_span<vertex_t> src,
                 raft::device_span<vertex_t> dst,
                 std::optional<raft::device_span<weight_t>> wgt,
                 size_t number_of_vertices,
                 int k)
{
  
  
  if (wgt.has_value()) {
    return detail::weighted_ktruss_subgraph_impl_(handle, src, dst, wgt, number_of_vertices, k);
  } else {
    //auto x =  detail::ktruss_subgraph_impl_(handle, src, dst, number_of_vertices, k);
    auto [result_src, result_dst] = detail::ktruss_subgraph_impl_(handle, src, dst, number_of_vertices, k);
    std::optional<rmm::device_uvector<weight_t>> result_wgt{std::nullopt};
    return std::make_tuple(std::move(result_src),
                         std::move(result_dst),
                         std::move(result_wgt));
  }

}

template std::tuple<rmm::device_uvector<int32_t>,
                    rmm::device_uvector<int32_t>,
                    std::optional<rmm::device_uvector<float>>>
k_truss_subgraph(raft::handle_t const& handle,
                 raft::device_span<int32_t> src,
                 raft::device_span<int32_t> dst,
                 std::optional<raft::device_span<float>> wgt,
                 size_t number_of_vertices,
                 int k);

template std::tuple<rmm::device_uvector<int32_t>,
                    rmm::device_uvector<int32_t>,
                    std::optional<rmm::device_uvector<double>>>
k_truss_subgraph(raft::handle_t const& handle,
                 raft::device_span<int32_t> src,
                 raft::device_span<int32_t> dst,
                 std::optional<raft::device_span<double>> wgt,
                 size_t number_of_vertices,
                 int k);

}  // namespace cugraph
